#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"

/*
 * In CUDA the codes can be ran on device (GPU) or host (GPU)
 * the __global__ qualifer defines that the function (kernel) should be compiled to run on device instead of the host
 */
__global__ void kernel(int a, int b, int *c) //in this function we define what will run on device
{
	*c = a + b;
} 

int main(void)
{
	int a = 10, b = 10, host_c;	//variables used by host
	int *device_c;	//variable used by device

	/*
	 * Allocates memory directly on device
	 * argument 1: pointer to the pointer to the memory block that you want to allocate
	 * argument 2: block's size that you want to allocate
	 */
	HANDLE_ERROR(hipMalloc((void**) &device_c, sizeof(int)));

	/*
	 * We call the kernel function and we pass some arguments for function using angle brackets <<<threads, gpu_blocks>>>
	 * argument 1: variable a (value copy)
	 * argument 2: variable b (value copy)
	 * argument 3: variable device_c (pointer to the memory block allocated on device)
	 */
	kernel<<<1,1>>>(a, b, device_c);

	/*
	 * Copy the value of device_c into host_c - data transfer of the DEVICE to HOST
	 * argument 1: pointer to the memory block on host mememory (destination)
	 * argument 2: pointer to the memory block on device memory (source)
	 * argument 3: block's size that you want to transfer
	 * argument 4: instruction that defines the sense of the memory transfer - variations: 
				   hipMemcpyDeviceToHost, 
				   hipMemcpyHostToDevice, 
				   hipMemcpyDeviceToDevice
	 */	
	HANDLE_ERROR(hipMemcpy(&host_c, device_c, sizeof(int), hipMemcpyDeviceToHost));

	printf("%d + %d = %d\n", a, b, host_c);
	
	hipFree(device_c);	//frees memory used on device

	return 0;
}
